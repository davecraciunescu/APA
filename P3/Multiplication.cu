#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstddef>
#include <iostream>

/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
* wA is A's width and wB is B's width.
*/
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA(float* C, float* A, float* B, int wA, int wB)
{
    // Indexes of the first sub-matrices processed by the block.
    int aBegin  = wA * BLOCK_SIZE * blockIdx.y;
    int bBegin  = BLOCK_SIZE * blockIdx.x;

    // Step sizes used to iterate through the different matrices.
    int aStep   = BLOCK_SIZE;
    int bStep   = BLOCK_SIZE * wB;

    // Indexes of the last sub-matrices procesed by the block.
    int aEnd    = aBegin + wA - 1;

    // Stores the element of the block sub-matrix computed by the thread.
    float Csub  = 0;

    // ------------------------------------------------------------------------

    // Compute the block sub-matrix.
    for (int a = aBegin, b = bBegin;
        a <= aEnd;
        a += aStep, b += bStep)
    {
        // Shared memory arrays used to store the sub-matrices.
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory to shared memory.
        As[threadIdx.y][threadIdx.x] = A[a + (wA * threadIdx.y) + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[b + (wB * threadIdx.y) + threadIdx.x];

        // Sync to make sure the matrices are loaded.
        __syncthreads();

        // Multiply the two matrices together.
        #pragma unroll // -> Optimize loop execution.
        for (int k = 0; k < BLOCK_SIZE, k++)
        {
            Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // Sync before loading new data.
        __syncthreads();
    }

    // ------------------------------------------------------------------------

    // Write to device memory.

    int c = (wB * BLOCK_SIZE * blockIdx.y) + (BLOCK_SIZE * blockIdx.x);
    C[c + (wB * threadIdx.y) + threadIdx.x] = Csub;
}

/**
* Data Initializer.
*/
void constantInit (float* data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/**
* Run a simple test of CUDA matrix multiplication.
*/
int matrixMultiply (int argc, char** argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices.
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);

    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float* h_C = (float*) malloc(mem_size_C);

    // Initiate host memory.
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, 0.01f);

    // Allocate device memory.
    float *d_A, *d_B, *d_C;

    // ------------------------------------------------------------------------

    if (h_C == nullptr)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    // ------------------------------------------------------------------------

    hipError_t error;

    // Allocate CUDA Memory for A.
    error = hipMalloc((void**) &d_A, mem_size_A);
    if (error != hipSuccess)
    {
        cudaMallocError("d_A", error);
    }

    // Allocate CUDA Memory for B.
    error = hipMalloc((void**) &d_B, mem_size_B);
    if (error != hipSuccess)
    {
        cudaMallocError("d_B", error);
    }

    // Allocate CUDA Memory for C.
    error = hipMalloc((void**) &d_C, mem_size_C);
    if (error != hipSuccess)
    {
        cudaMallocError("d_C", error);
    }

    // ------------------------------------------------------------------------
    
    // Copy host memory to device.
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        cudaMemcpyError("d_A", "h_A", error);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        cudaMemcpyError("d_B", "h_B", error);
    }

    // ------------------------------------------------------------------------

    // Setup execution parameters.
    dim3 threads(block_size, block_size);
    dim3 grid( (dimsB.x / threads.x), (dimsA.y / threads.y) );

    // ------------------------------------------------------------------------

    // Execute kernel.
    int nIter = 300;

    for (int j = 0; j < nIter; j++)
    {
        matrixMulCUDA<16> <<<grid, threads>>> (d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    // ------------------------------------------------------------------------

    // Copy result from device to host.
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        cudaMemcpyError("h_C", "d_C", error);
    }

    // ------------------------------------------------------------------------

    // Clean up host memory.
    free(h_A);
    free(h_B);
    free(h_C);

    // Clean up device memory.
    error = hipFree(d_A);
    if (error != hipSuccess)
    {
        cudaFreeError("d_A", error);
    }

    error = hipFree(d_B);
    if (error != hipSuccess)
    {
        cudaFreeError("d_B", error);
    }

    error = hipFree(d_C);
    if (error != hipSuccess)
    {
        cudaFreeError("d_C", error);
    }
}

/**
* Program Main.
*/
int main (int argc, char **argv)
{
    // PRINT STARTING MESSAGE:

    int block_size = 32;

    dim3 dimsA(16, 16, 1);
    dim3 dimsB(16, 16, 1);

    // Set up arbitrary parameters.
    dimsA.x = 16;
    dimsA.y = 16;

    dimsB.x = 16;
    dimsB.y = 16;

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}

// Error Handling Methods.
void cudaFreeError(std::string id, hipError_t msg)
{
    std::cout <<"Failed to free device from "<< id <<" (error code "<<
        hipGetErrorString(msg) << ")!\n";
    exit(EXIT_FAILURE);
}

void cudaMemcpyError(std::string from, std::string to, hipError_t msg)
{
    std::cout <<"hipMemcpy ("<< from << to <<") returned error "<<
    hipGetErrorString(msg) <<"(code "<< error <<"), line ("<<__LINE__ <<")\n";
    exit(EXIT_FAILURE);
}

void cudaMallocError(std::string id, hipError_t msg)
{
    std::cout <<"hipMalloc "<< id <<"returned error "<< hipGetErrorString(msg)
    <<" (code "<< error <<"), line ("<<__LINE__<<")\n";
    exit(EXIT_FAILURE);
}
