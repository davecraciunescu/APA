#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>


hipError_t multMatCuda(int* result, int* a, int* b, int size);

__global__ void multMat(int* result, int* a, int* b, int size)
{
	// Identifier of the corresponding executino at the GPU. It will be
    // obtained from the block's ID and the corresponging thread's ID
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < size && row < size)
    {
        for(int i = 0; i < size; i++)
        {
            result[row * size + col] += a[col * size + i] * b[i * size + col];
        }
    }
}

int main()
{
    const int size = 16;

	int* a      = (int*)calloc(size * size, sizeof(int));  
	int* b      = (int*)calloc(size * size, sizeof(int));  
	int* result = (int*)calloc(size * size, sizeof(int));  
    
    // Data loaded into the arrays
    for(int i = 0; i < size; i++)
    {
        for(int j = 0; j < size; j++)
        {
            a[i * size + j] = 1;
            b[i * size + j] = 2;
        }
    }

    printf("Data Initialized\n");

    // Kernel method execution
    hipError_t err = multMatCuda(result, a, b, size);
	if (err != hipSuccess) {
		fprintf(stderr, "multMat failed!\n");
		return 1;
	}
    
    // Operation is printed
    for(int i = 0; i < size; i++)
    {
        for(int j = 0; j < 3; j++)
        {
            printf("| ");
            
            for(int k = 0; k < size; k++)
            {
                switch(j)
                {
                    case 0:
                        printf("%d", a[i * size + k]);
                        break;
                    
                    case 1:
                        printf("%d", b[i * size + k]);
                        break;
                
                    case 2:
                        printf("%d", result[i * size + k]);
                        break;
                }
                if(k != size - 1) printf(", ");
            }

            if(i != (int) (size / 2) && j != 2) {
                printf(" |   ");
            } else if(j == 2) {
                printf(" |");
            } else {
                switch(j)
                {
                    case 0:
                        printf(" | * ");
                        break;
                    
                    case 1:
                        printf(" | = ");
                        break;
                    case 2:
                        printf(" |");
                }
            }

        }

        printf("\n");
    }

	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}

hipError_t multMatCuda(int* result, int* a, int* b, int size)
{
    int TILE_WIDTH = 1;
	// As many cuda errors are going to be treated, it is better to initialize
	// here the error variable
	hipError_t err;
	// Variables used to operate in GPU mem
	int* dev_result = 0;
	int* dev_a      = 0;
	int* dev_b      = 0;

    dim3 dimGrid(size / TILE_WIDTH, size / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	
    // Select GPU where the threads are to be executed
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!");
		goto Error;
	}

	// Allocate memory for GPU's variables
	err = hipMalloc((void**)&dev_result, size * size * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_a, size * size * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_b, size * size * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	// Transfers data from host variables to GPU variables
	err = hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	err = hipMemcpy(dev_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


    // Launch kernel to multiply 2 16x16 matrices
	multMat<<<dimGrid, dimBlock>>>(dev_result, dev_a, dev_b, size);

	// Checks whether the execution in the GPU has been completed correctly
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "multMat launch failed: %s\n",
			hipGetErrorString(err));
		goto Error;
	}

	// Waits for the kernel to finish and checks whether theres has been any
	// errors or not
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching multMat!\n", err);
		goto Error;
	}

	err = hipMemcpy(result, dev_result, size * size * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_a);
	hipFree(dev_b);

	return err;
}

