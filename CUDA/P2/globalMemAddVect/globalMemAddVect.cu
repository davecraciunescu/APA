#include <hip/hip_runtime.h>

#include <stdio.h>

hipError_t addVectCuda(int* result, int* a, int* b, int size);

__global__ void addVect(int* result, int* a, int* b, int size)
{
	// Identifier of the corresponding executino at the GPU. It will be
    // obtained from the block's ID and the corresponging thread's ID
    int pos = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(pos < size) result[pos] = a[pos] + b[pos];
}

int main()
{
    const int size = 24;

	int a[size];  
	int b[size]; 
	int result[size];
    
    // Data loaded into the arrays
    for(int i = 0; i < size; i++)
    {
        a[i] = -i;
        b[i] = i + i;
    }
	
    // Kernel method execution
    hipError_t err = addVectCuda(result, a, b, size);
	if (err != hipSuccess) {
		fprintf(stderr, "addVectCuda failed!\n");
		return 1;
	}
    
    // Operation is printed
    // Print A
    printf("  {");
    for(int i = 0; i < size; i++)
    {
        printf("%d", a[i]);
        if(i != size - 1) printf(", ");
    }
    printf("}\n+ {");
    // Print B
    for(int i = 0; i < size; i++)
    {
        printf("%d", b[i]);
        if(i != size - 1) printf(", ");
    }
    printf("}\n");
    printf("---------------------------------------------------------------------------------------------");
    printf("\n  {");
    // Print Result
    for(int i = 0; i < size; i++)
    {
        printf("%d", result[i]);
        if(i != size - 1) printf(", ");
    }
    printf("}");

	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}

hipError_t addVectCuda(int* result, int* a, int* b, int size)
{
	// As many cuda errors are going to be treated, it is better to initialize
	// here the error variable
	hipError_t err;
	// Variables used to operate in GPU mem
	int* dev_result = 0;
	int* dev_a = 0;
	int* dev_b = 0;

	// Select GPU where the threads are to be executed
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!");
		goto Error;
	}

	// Allocate memory for GPU's variables
	err = hipMalloc((void**)&dev_result, size * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	// Transfers data from host variables to GPU variables
	err = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	err = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch kernel to add three values
	addVect <<<3, 8>>>(dev_result, dev_a, dev_b, size);

	// Checks whether the execution in the GPU has been completed correctly
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "addVect launch failed: %s\n",
			hipGetErrorString(err));
		goto Error;
	}

	// Waits for the kernel to finish and checks whether theres has been any
	// errors or not
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addThreeNums!\n", err);
		goto Error;
	}

	err = hipMemcpy(result, dev_result, size * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_a);
	hipFree(dev_b);

	return err;
}

