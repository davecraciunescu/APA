#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

// Sets the size of the tile to be used
#define TILE_WIDTH 2 

/* 
 * Method which tests whether there occur erroes while doing the operations to
 * move data from CPU to GPU and viceversa
 */
hipError_t multMatCuda(int* result, int* a, int* b, int size);

__global__ void multMat(int* result, int* a, int* b, int size)
{
    // Elements which store elements in shared memory
	__shared__ int sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ int sB[TILE_WIDTH][TILE_WIDTH];
    // MAtrices dimensions
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    // Identifier of the corresponding executino at the GPU. It will be obtained
    // from the block's ID and the corresponging thread's ID
    int col = bx * TILE_WIDTH + tx; 
    int row = by * TILE_WIDTH + ty;
   
    float mult = 0.0;
    // Are the column and row computed within the rigth limits of the matrices?
    if(col < size && row < size)
    {
        for(int i = 0; i < size / TILE_WIDTH; ++i)
        {
            // Copy data from global mem. to shared mem.
            sA[ty][tx] = a[row * size + (i * TILE_WIDTH + tx)];
            sB[ty][tx] = b[(i * TILE_WIDTH + ty) * size + col];
            
            __syncthreads();

            // Multiplication
            for(int j = 0; j < TILE_WIDTH; ++j) {
                mult += sA[ty][j] * sB[j][tx];
            }

            __syncthreads();
        }

        result[row * size + col] = mult;
    }
}

/*
 * Method which simplifies CUDA's error treatment
 */
__host__ void check_CUDA_Error(const char *msg)
{
    hipError_t err;

    hipDeviceSynchronize();

    err = hipGetLastError();

    if(err != hipSuccess)
    {
        printf("ERROR %d OCURRED: %s (%s)\n", err, hipGetErrorString(err), msg);
        printf("Press any key to finish execution...");
        fflush(stdin);
        char tecla = getchar();
        exit(-1);
    }
}

int main()
{
    const int size = 16;

	int* a      = (int*)calloc(size * size, sizeof(int));  
	int* b      = (int*)calloc(size * size, sizeof(int));  
	int* result = (int*)calloc(size * size, sizeof(int));  
    
    // Data loaded into the arrays
    for(int i = 0; i < size; i++)
    {
        for(int j = 0; j < size; j++)
        {
            a[i * size + j] = 1;
            b[i * size + j] = 2;
        }
    }

    printf("Data Initialized\n");

    // Kernel method execution
    multMatCuda(result, a, b, size);
	check_CUDA_Error("The matrix multiplication failed at multMat!\n");

    printf("Multiplication completed.\nIt is recommended to use the screen");
    printf("maximized to watch correctly the operation.\n");

    // Operation is printed
    for(int i = 0; i < size; i++)
    {
        for(int j = 0; j < 3; j++)
        {
            printf("| ");
            
            for(int k = 0; k < size; k++)
            {
                switch(j)
                {
                    case 0: // Matrix A
                        printf("%d", a[i * size + k]);
                        break;
                    
                    case 1: // Matrix B
                        printf("%d", b[i * size + k]);
                        break;
                
                    case 2: // Matrix Result
                        printf("%d", result[i * size + k]);
                        break;
                }
                if(k != size - 1) printf(", ");
            }

            if(i != (int) (size / 2) && j != 2) {
                printf(" |   ");
            } else if(j == 2) {
                printf(" |");
            } else {
                switch(j)
                {
                    case 0:
                        printf(" | * ");
                        break;
                    
                    case 1:
                        printf(" | = ");
                        break;
                    case 2:
                        printf(" |");
                }
            }

        }

        printf("\n");
    }

	hipDeviceReset();
	check_CUDA_Error("hipDeviceReset failed!\n");

	return 0;
}

/* 
 * Method which tests whether there occur erroes while doing the operations to
 * move data from CPU to GPU and viceversa
 */
hipError_t multMatCuda(int* result, int* a, int* b, int size)
{
	// Variables used to operate in GPU mem
	int* dev_result = 0;
	int* dev_a      = 0;
	int* dev_b      = 0;

    dim3 dimGrid(size / TILE_WIDTH, size / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	
    // Select GPU where the threads are to be executed
	hipSetDevice(0);
	check_CUDA_Error("hipSetDevice failed!\n");

	// Allocate memory for GPU's variables
	hipMalloc((void**)&dev_result, size * size * sizeof(int));
	check_CUDA_Error("hipMalloc failed at Matrix Result!\n");

    hipMalloc((void**)&dev_a, size * size * sizeof(int));
	check_CUDA_Error("hipMalloc failed at Matrix A!\n");

	hipMalloc((void**)&dev_b, size * size * sizeof(int));
	check_CUDA_Error("hipMalloc failed at Matrix B!\n");

	// Transfers data from host variables to GPU variables
	hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
	check_CUDA_Error("hipMemcpy failed at Matrix A!\n");

	hipMemcpy(dev_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);
	check_CUDA_Error("hipMemcpy failed at Matrix B!");

    // Launch kernel to multiply 2 16x16 matrices
	multMat<<<dimGrid, dimBlock>>>(dev_result, dev_a, dev_b, size);

	// Checks whether the execution in the GPU has been completed correctly
    check_CUDA_Error("multMat launch failed");	

	// Waits for the kernel to finish and checks whether theres has been any
	// errors or not
	hipDeviceSynchronize();
	check_CUDA_Error("hipDeviceSynchronize returned error after launching multMat!");

	hipMemcpy(result, dev_result, size * size * sizeof(int), hipMemcpyDeviceToHost);
	check_CUDA_Error("hipMemcpy failed after copying the result from GPU to CPU!");

    return hipGetLastError();
}

