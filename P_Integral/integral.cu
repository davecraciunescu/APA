#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

// Numero de intervalos
const int N = 13500;
const int numThreads = 256;
const int bloquesPorGrid = min(32, (N + numThreads - 1) / numThreads);

__device__ float funcion_gpu(float x)
{
    return cos(pow(x, 2) / 2) * pow(x, 3) * sin(pow(x, 2)) / (pow(x, 2) + 3);
}

float funcion_cpu(float x)
{
    return cos(pow(x, 2) / 2) * pow(x, 3) * sin(pow(x, 2)) / (pow(x, 2) + 3);
}

__global__ void trapecios(float a, float b, float h, float* resultado)
{
    __shared__ float parcial[numThreads];
    int iteracion = threadIdx.x + blockIdx.x * blockDim.x;
    float temp = 0;

    while (iteracion < N)
    {
        if (iteracion != 0)
        {
            temp += funcion_gpu(a + h * iteracion);
        }

        iteracion += blockDim.x * gridDim.x;
    }

    parcial[threadIdx.x] = temp; // Almacena los resultados parciales

    __syncthreads(); // Sincroniza threads

    int i = blockDim.x / 2;

    while (i != 0) // Fase de reduccion
    {
        if (threadIdx.x < i)
            parcial[threadIdx.x] += parcial[threadIdx.x + i];

        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0)
        resultado[blockIdx.x] = parcial[0];
}

int main(int argc, char **argv)
{    
    const float a = 1.5f;
    const float b = 2.78f;
    
    float h = (b - a) / N;
    float* resultado_cpu;
    float* resultado_gpu;
    
    resultado_cpu = (float*) malloc(bloquesPorGrid * sizeof(float));
    
    hipError_t err = hipMalloc(&resultado_gpu, bloquesPorGrid *  sizeof(float));
    
    trapecios<<< bloquesPorGrid,numThreads >>> (a, b, h, resultado_gpu);
    
    err = hipMemcpy(resultado_cpu, resultado_gpu,
                     bloquesPorGrid * sizeof(float), 
                     hipMemcpyDeviceToHost);

    float suma_parciales = (funcion_cpu(a) + funcion_cpu(b)) / 2.0f;

    for (int i = 0; i < bloquesPorGrid; i++)
    {
        suma_parciales += resultado_cpu[i];
    }

    suma_parciales *= h;
    
    std::cout << "Resultado de integral con GPU: " << suma_parciales << std::endl;
    suma_parciales = (funcion_cpu(a) + funcion_cpu(b)) / 2.0f;
    
    for(int i=1; i < N; i++)
    {
        suma_parciales += funcion_cpu(a + i*h);
    }

    suma_parciales *= h;
    std::cout << "Resultado de integral con CPU: " << suma_parciales << std::endl;
    hipFree(resultado_gpu);
    free(resultado_cpu);
    return 0;
}

