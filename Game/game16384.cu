// -----------------------------------------------------------------------------
// ------------------------------ LIBRARIES ------------------------------------
// -----------------------------------------------------------------------------
// Includes CUDA
#include <hip/hip_runtime.h>
// CUDA get characteristics

// Allows to implement all string related methods and variables
#include <string>
// C++ library for I/O 
#include <iostream>
// Calloc, exit, free
#include <stdlib.h>
// Rand
#include <cstdlib>
// Time
#include <time.h>
// Slepp
#include <chrono>
#include <thread>

// Size of the tile to be used
#define TILE_WIDTH 1

// -----------------------------------------------------------------------------
// ------------------------------- HEADERS -------------------------------------
// -----------------------------------------------------------------------------
hipError_t cellsMerge(char movement, int row, int column, int* matrix,
                       int* POINTS, int* CELLS_OCCUPIED);

// -----------------------------------------------------------------------------
// ------------------------------- KERNELS -------------------------------------
// -----------------------------------------------------------------------------
__global__ void computeMatrixUp(int numRows, int numColumns, int* matrix,
                                int* POINTS, int* CELLS_OCCUPIED)
{
    // Matrix dimensions
    int bx = blockIdx.x;
    int tx = threadIdx.x;

    // Location in matrix
    int col = bx * TILE_WIDTH + tx;

    if(col < numColumns)
    {
        // TODO: Probably will need to change the numRows if TILES or shared 
        // used
        for(int i = 0; i < numRows - 1; i++)
        {
            if(matrix[i * numRows + col] > 0 && 
               matrix[i * numRows + col] == matrix[(i + 1) * numRows + col])
            {
                matrix[i * numRows + col] *= 2;
                matrix[(i + 1) * numRows + col] = 0;
                (*POINTS) += matrix[i * numRows + col];
                (*CELLS_OCCUPIED)--;
            }
        }
    }
}

__global__ void computeMatrixDown(int numRows, int numColumns, int* matrix,
                                  int* POINTS, int* CELLS_OCCUPIED)
{
    // Matrix dimensions
    int bx = blockIdx.x;  
    int tx = threadIdx.x;

    // Location in matrix
    int col = bx * TILE_WIDTH + tx;

    if(col < numColumns)
    {
        // TODO: Probably will need to change the numRows if TILES or shared 
        // used
        for(int i = numRows - 1; i > 0; i--)
        {
            if(matrix[i * numRows + col] > 0 && 
               matrix[i * numRows + col] == matrix[(i - 1) * numRows + col])
            {
                matrix[i * numRows + col] *= 2;
                matrix[(i - 1) * numRows + col] = 0;
                (*POINTS) += matrix[i * numRows + col];
                (*CELLS_OCCUPIED)--;
            }
        }
    }
}

__global__ void computeMatrixLeft(int numRows, int numColumns, int* matrix,
                                  int* POINTS, int* CELLS_OCCUPIED)
{
    // Matrix dimensions
    int bx = blockIdx.x;
    int tx = threadIdx.x;

    // Location in matrix
    int row = bx * TILE_WIDTH + tx;

    if(row < numRows)
    {
        // TODO: Probably will need to change the numRows if TILES or shared 
        // used
        for(int i = 0; i < numColumns - 1; i++)
        {
            if(matrix[row * numRows + i] > 0 && 
               matrix[row * numRows + i] == matrix[row * numRows + (i + 1)])
            {
                matrix[row * numRows + i] *= 2;
                matrix[row * numRows + (i + 1)] = 0;
                (*POINTS) += matrix[row * numRows + i];
                (*CELLS_OCCUPIED)--;
            }
        }
    }
}

__global__ void computeMatrixRight(int numRows, int numColumns, int* matrix,
                                   int* POINTS, int* CELLS_OCCUPIED)
{
    // Matrix dimensions
    int bx = blockIdx.x;  
    int tx = threadIdx.x;

    // Location in matrix
    int row = bx * TILE_WIDTH + tx;

    if(row < numRows)
    {
        // TODO: Probably will need to change the numRows if TILES or shared 
        // used
        for(int i = numColumns - 1; i > 0; i--)
        {
            if(matrix[row * numRows + i] > 0 && 
               matrix[row * numRows + i] == matrix[row * numRows + (i - 1)])
            {
                matrix[row * numRows + i] *= 2;
                matrix[row * numRows + (i - 1)] = 0;
                (*POINTS) += matrix[row * numRows + i];
                (*CELLS_OCCUPIED)--;
            }
        }
    }
}

/*
 * Fills the empty spaces in the matrix
 *
 * matrix: The matrix which is going to be filled
 * movement: Movement performed by the user (w, s, a, d)
 */
__global__ void fillSpace(int* matrix, char movement, int rows, int columns)
{
    // Matrix dimensions
    int bx = blockIdx.x;  
    int tx = threadIdx.x; 

    // Location in matrix
    int pos = bx * TILE_WIDTH + tx;
    
    switch(movement)
    {
        // Up
        case 'w':
            // From last row to first row
            for(int i = rows - 1; i > 0; i--)
            {
                for(int j = rows - 1; j > 0; j--)
                {
                    // Current cell NOT 0 and upper cell IS 0 ->
                    // moves current cell up
                    if(matrix[j *     rows + pos] != 0 &&
                       matrix[(j - 1) * rows + pos] == 0)
                    {
                        matrix[(j - 1) * rows + pos] = matrix[j * rows + pos];
                        matrix[j * rows + pos] = 0;
                    }
                }
            }
            break;

        // Down
        case 's':
            // From first row to last row
            for(int i = 0; i < rows; i++)
            {
                for(int j = 0; j < rows - 1; j++)
                {
                    // Current cell NOT 0 and lower cell IS 0 ->
                    // moves current cell down
                    if(matrix[j *     rows + pos] != 0 &&
                       matrix[(j + 1) * rows + pos] == 0)
                    {
                        matrix[(j + 1) * rows + pos] = matrix[j * rows + pos];
                        matrix[j * rows + pos] = 0;
                    }
                }
            }
            break;

        // Left
        case 'a':
            // From last row to first row
            for(int i = columns - 1; i > 0; i--)
            {
                for(int j = columns - 1; j > 0; j--)
                {
                    // Current cell NOT 0 and upper cell IS 0 ->
                    // moves current cell up
                    if(matrix[pos * rows +  j]      != 0 &&
                       matrix[pos * rows + (j - 1)] == 0)
                    {
                        matrix[pos * rows + (j - 1)] = matrix[pos * rows + j];
                        matrix[pos * rows +  j] = 0;
                    }
                }
            }
            
            break;

        //  Right
        case 'd':
            // From first row to last row
            for(int i = 0; i < columns; i++)
            {
                for(int j = 0; j < columns - 1; j++)
                {
                    // Current cell NOT 0 and lower cell IS 0 ->
                    // moves current cell down
                    if(matrix[pos * rows + j]       != 0 &&
                       matrix[pos * rows + (j + 1)] == 0)
                    {
                        matrix[pos * rows + (j + 1)] = matrix[pos * rows + j];
                        matrix[pos * rows +  j] = 0;
                    }
                }
            }
            break;
    }

}

// -----------------------------------------------------------------------------
// ------------------------- FUNCTIONALITY METHODS  ----------------------------
// -----------------------------------------------------------------------------
// Method which allows to check for errors
__host__ void check_CUDA_Error(const char *msg)
{
    hipError_t  err;

    hipDeviceSynchronize();

    err = hipGetLastError();

    if(err != hipSuccess)
    {
        std::cout << "ERROR " << err << " OCURRED: " << hipGetErrorString(err)  
                  << "(" << msg << ")" << std::endl;
        std::cout << "Press any key to finish execution..." << std::endl;
        fflush(stdin);

        char key = getchar();

        exit(-1);
    }
}

// Gets the number of threads allowed per block in the current GPU
__host__ int getThreadsBlock()
{
    int threadsBlock = 0;

    int nDevices;

    hipGetDeviceCount(&nDevices);
    check_CUDA_Error("Couldn't get the number of devices in this computer");

    if(nDevices > 0)
    {
        hipDeviceProp_t prop;
        
        hipGetDeviceProperties(&prop, 0);
        
        threadsBlock = prop.maxThreadsPerBlock;
    }

    return threadsBlock;
}

// -----------------------------------------------------------------------------
// ------------------------------ GAME METHODS ---------------------------------
// -----------------------------------------------------------------------------
__host__ std::string printHearts(int* LIVES)
{
    std::string hearts;

    for(int i = 0; i < *LIVES; i++) {
        hearts += "<3 ";
    }

    return hearts;
}

__host__ void displayGrid(int rows, int columns, int* Matrix, 
                          int* POINTS, int* LIVES, int* CELLS_OCCUPIED)
{
    system("clear");

    // Two extra iterations to print the upper part of the matrix
    for(int i = -2; i < rows; i++)
    {
        if(i < 0) {
            std::cout << "      ";
        } else if(i + 1 < 10) {
            std::cout << i + 1 << " - ";
        } else if(i + 1 >= 10) {
            std::cout << i + 1 << "- ";
        }

        for(int j = 0; j < columns; j++)
        {
            if(i == -2) {
                if(j + 1 < 10) {
                    std::cout << j + 1 << "    ";
                } else {
                    std::cout << j + 1 << "   ";
                }
            } else if(i == -1) {
                std::cout << "|    "; 
            } else  {

                switch(Matrix[i * rows + j])
                {
                    // LIGHTWHITE
                    case 2:
                        std::cout << "\033[1;37;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;
 
                    // WHITE
                    case 4:
                        std::cout << "\033[1;37m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;
                    
                    // DARKGRAY
                    case 8:
                        std::cout << "\033[1;30;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;
                    
                    // YELLOW
                    case 16:
                        std::cout << "\033[1;33m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // LIGHTMAGENTA
                    case 32:
                        std::cout << "\033[1;35;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;
                    
                    // MAGENTA
                    case 64:
                        std::cout << "\033[1;35m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // LIGHTRED
                    case 128:
                        std::cout << "\033[1;31;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // RED
                    case 256:
                        std::cout << "\033[1;31m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // TODO
                    // BROWN
                    case 512:
                        std::cout << "\033[1;37m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // LIGHTGREEN
                    case 1024:
                        std::cout << "\033[1;32;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // GREEN
                    case 2048:
                        std::cout << "\033[1;32m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;
                    
                    // LIGHTCYAN
                    case 4096:
                        std::cout << "\033[1;36;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // CYAN
                    case 8192:
                        std::cout << "\033[1;36m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    // LIGHTBLUE
                    case 16384:
                        std::cout << "\033[1;34;1m| " << Matrix[i * rows + j] 
                                  << " |\033[0m";
                        break;

                    default:
                        std::cout << "| " << Matrix[i * rows + j] << " |";
                        
                        break;
                }
            }
        }

        std::cout << std::endl;
    }

    std::cout << std::endl << std::endl << std::endl 
              << "Controls:            Save    Quit     Points:"
              << "        Cells           Lives:"                   << std::endl
              << "        ___                           "
              << "               Occupied:"                         << std::endl
              << "       | W |          ___     ___     "           << std::endl
              << " ___    ___    ___   | G |   | Q |    "           << *POINTS 
              << "              " << *CELLS_OCCUPIED                << std::endl
              << "| A |  | S |  | D |                                          "
              << "        " << printHearts(LIVES) 
              << std::endl << std::endl;
}

__host__ void seeding(int gameDifficulty, int rows, int columns, int* matrix,
                      int* CELLS_OCCUPIED)
{
    // Number of seeds to be planted in the board
    int seeds;
    // Values the seeds might have while inserted
    int* seedsValues;
    // Number of seeds planted 
    int seedsPlanted = 0;
    // Position from the matrix where the seed is going to be planted. 
    // Used auxiliary variable
    int position;

    // Depending on the game difficulty, the number of seeds may vary
    switch(gameDifficulty)
    {
        case 1:
            seeds = 15;
            seedsValues = (int*) calloc(3, sizeof(int));
            seedsValues[0] = 2;
            seedsValues[1] = 4;
            seedsValues[2] = 8;
            break;

        case 2:
            seeds = 8;
            seedsValues = (int*) calloc(2, sizeof(int));
            seedsValues[0] = 2;
            seedsValues[1] = 4;
            break;
    }

    // Initialize random seed
    std::srand(time(NULL));

    std::cout << "CELLS_OCCUPIED: " << (*CELLS_OCCUPIED) << std::endl;

    while(seedsPlanted < seeds)
    {
        // Still empty cells
        if((*CELLS_OCCUPIED) <= (rows * columns))
        {
            // Position within the matrix
            position = rand() % (rows * columns);

            if(matrix[position] == 0)
            {
                // Random seed value among the ones according to the difficulty
                matrix[position] = seedsValues[rand() % 
                                               (sizeof(seedsValues) 
                                                / sizeof(int))];
                seedsPlanted++;
                (*CELLS_OCCUPIED)++;     
            }
        } 
        else
        {
            std::cout << "There are no more empty cells, you have lost a live"  << std::endl;
            // TODO:
            // Probably it would be needed decrease the number of lives
            // Therefore, it is needed a new method for that, called here
        }

    }

}

/**
* Asks user if will play again.
*/
bool playAgain(int lives)
{
    bool willPlayAgain = false;

    std::cout << "You currently have: " << lives << " lives." << std::endl;
    std::cout << "Do you want to play again (y/n).";

    std::string input;
    std::cin >> input;

    bool invalid = true;

    while (invalid)
    {
        if (input.length() == 1)
        {
            switch(input[0])
            {
                case 'y':
                    std::cout << "Alright, playing again." << std::endl;
                    willPlayAgain   = true;
                    invalid         = false;
                    break;
                
                case 'n':
                    std::cout << "Thanks for playing." << std::endl;
                    invalid         = false;
                    break;
                
                default:
                    std::cout << "Please enter a valid value." << std::endl;
                    std::cout << "Do you want to play again (y/n).";
                    break;
            }
        }
        else
        {
            std::cout << "Please enter a valid value." << std::endl;
        }
    }
    return willPlayAgain;
}

void playGameManual (
    int  difficulty,    // Difficulty of the game.
    int  numRows,       // Number of rows in the game.
    int  numColumns,    // Number of columns in the game.
    int  numMaxThreads  // Number of max threads to be run.
    )
{

    // Auxiliary input variable.
    std::string input;

    // Variables needed within the game.
    int   lives = 5;    int* LIVES          = &lives;
    int  points = 0;    int* POINTS         = &points;
    int cellsOc = 0;    int* CELLS_OCCUPIED = &cellsOc; 

    int*       matrix = (int*) calloc(numRows * numColumns, sizeof(int)); 
    bool      playing = true;
    bool  keepPlaying = true;
    bool       winner = true;

    // MAIN GAME LOOP.
    while (keepPlaying)
    {   
        // SEED GAME INITIAL STATE.
        seeding(difficulty, numRows, numColumns, matrix, CELLS_OCCUPIED);
        displayGrid(numRows, numColumns, matrix, POINTS, LIVES, CELLS_OCCUPIED);

        std::cout << "Starting Game." << std::endl;

        if (lives > 0)
        {
            while (playing)
            {
                std::cin >> input;

                if (input.length() == 1)
                {
                    switch (input[0])
                    {
                        case 'g':
                            // TODO SAVE GAME.
                        break;
                        
                        case 'q':
                            playing        = false;
                            keepPlaying    = false;
                            winner         = false;
                        break;
                        
                        default:
                            if (winner) // TODO: CONNECT WINNER WITH KERNEL.
                            {
                                cellsMerge(input[0], numRows, numColumns, matrix,
                                    POINTS, CELLS_OCCUPIED);
                                seeding(difficulty, numRows, numColumns, matrix,
                                    CELLS_OCCUPIED);
                                displayGrid(numRows, numColumns, matrix, POINTS, LIVES,
                                    CELLS_OCCUPIED);
                            }
                            else
                            {
                                // Take away one life.
                                lives--;
                                playing = false;
                            }
                        break;
                    }
                }
                else
                {
                    std:: cout << "Not that one, cracker!" << std::endl;
                }
            }

            // ASK USER IF WANTS TO PLAY AGAIN.
            keepPlaying = playAgain(lives);
        }
        else
        {
            std::cout << "You have 0 lives. GAMEOVER." << std::endl;
            keepPlaying = false;
        }
    }

    // Reset the value of winner.
    winner = true;
}

void playGameAutomatic (
    int  difficulty,    // Difficulty of the game.
    int  numRows,       // Number of rows in the game.
    int  numColumns,    // Number of columns in the game.
    int  numMaxThreads  // Number of max threads to be run.
    )
{

    // Auxiliary input variable.
    std::string input;

    // Variables needed within the game.
    int   lives = 5;    int* LIVES          = &lives;
    int  points = 0;    int* POINTS         = &points;
    int cellsOc = 0;    int* CELLS_OCCUPIED = &cellsOc; 

    int*       matrix = (int*) calloc(numRows * numColumns, sizeof(int)); 
    bool      playing = true;
    bool  keepPlaying = true;
    bool       winner = true;

    char movements [4] = {'w', 'a', 's', 'd'};

    // Initialize random seed
    std::srand(time(NULL));

    /* Used to know when to ask the user for an action:
     * - Continue.
     * - Quit.
     * - Save.
     */
    int iteration = 0;

    // MAIN GAME LOOP.
    while (keepPlaying)
    {   
        // SEED GAME INITIAL STATE.
        seeding(difficulty, numRows, numColumns, matrix, CELLS_OCCUPIED);
        displayGrid(numRows, numColumns, matrix, POINTS, LIVES, CELLS_OCCUPIED);

        std::cout << "Starting Game." << std::endl;

        if (lives > 0)
        {
            while (playing)
            {   

                std::this_thread::sleep_for(std::chrono::milliseconds(1200));

                if(iteration % 10 == 0)
                {
                    std::cout << "Do you wish to SAVE (G) your game? "
                              << std::endl
                              << "Or maybe to QUIT (Q) the game?"
                              << std::endl
                              << "If you want to keep playing in the "
                              << "automatical mode PRESS ANY KEY."
                              << std::endl;

                    std::cin >> input;
                    iteration = 1;
                    
                    if (input.length() == 1)
                    {
                        switch (input[0])
                        {
                            case 'g':
                                // TODO SAVE GAME.
                            break;
                            
                            case 'q':
                                playing        = false;
                                keepPlaying    = false;
                                winner         = false;
                            break;
                            
                            default:
                                if (winner) // TODO: CONNECT WINNER WITH KERNEL.
                                {
                                    cellsMerge(input[0], numRows, numColumns, matrix,
                                        POINTS, CELLS_OCCUPIED);
                                    seeding(difficulty, numRows, numColumns, matrix,
                                        CELLS_OCCUPIED);
                                    displayGrid(numRows, numColumns, matrix, POINTS, LIVES,
                                        CELLS_OCCUPIED);
                                }
                                else
                                {
                                    // Take away one life.
                                    lives--;
                                    playing = false;
                                }
                            break;
                        }
                    }
                    else
                    {
                        std:: cout << "Not that one, cracker!" << std::endl;
                    }
                } 
                else 
                {
                    iteration++;

                    if (winner) // TODO: CONNECT WINNER WITH KERNEL.
                    {
                        int movement = rand() %
                                       (sizeof(movements) / sizeof(char));
                        cellsMerge(movements[movement], numRows, numColumns, matrix,
                                   POINTS, CELLS_OCCUPIED);
                        seeding(difficulty, numRows, numColumns, matrix,
                                CELLS_OCCUPIED);
                        displayGrid(numRows, numColumns, matrix, POINTS, LIVES,
                                    CELLS_OCCUPIED);
                    }
                    else
                    {
                        // Take away one life.
                        lives--;
                        playing = false;
                    }
                }

            }

            // ASK USER IF WANTS TO PLAY AGAIN.
            keepPlaying = playAgain(lives);
        }
        else
        {
            std::cout << "You have 0 lives. GAMEOVER." << std::endl;
            keepPlaying = false;
        }
    }

    // Reset the value of winner.
    winner = true;
    
}

// MAIN METHOD OF THE GAME.
void playGame (
    int  difficulty,    // Difficulty of the game.
    int  numRows,       // Number of rows in the game.
    int  numColumns,    // Number of columns in the game.
    int  numMaxThreads, // Number of max threads to be run.
    char mode           // Gaming mode (manual or automatic).
    )
{
    switch(mode)
    {
        case 'm':
            playGameManual(difficulty, numRows, numColumns, numMaxThreads);
            break;

        case 'a':
            playGameAutomatic(difficulty, numRows, numColumns, numMaxThreads);
            break;
    }
}

// -----------------------------------------------------------------------------
// -------------------------------- MAIN CODE ----------------------------------
// -----------------------------------------------------------------------------
int main(int argc, char** argv)
{
    // Variables needed as game settings.
    char mode;          // Game mode.
    int  difficulty;    // Difficulty of the game.
    int  numRows;       // Number of rows in the game.
    int  numColumns;    // Number of columns in the game.
    int  numMaxThreads; // Number of max threads to be run.
    
    // Used as auxiliary variable for any input in the system
    std::string input;
    

    system("clear");
    std::cout << "Processing game settings" << std::endl;

    if(argc == 5)
    {
        input = argv[1];
        
        // --------------------------------------------------------------------
        // --------------------------- PLAYING MODE ---------------------------
        // --------------------------------------------------------------------
        if(input.compare("m") == 0) 
        {
            std::cout << "Setting to MANUAL" << std::endl;   
            std::cout << "-------------------------------" 
                      << " MANUAL MODE COMPLETED 100% " 
                      << "-------------------------------"
                      << std::endl;
            mode = 'm';
        } else if (input.compare("a") == 0)
        {
            std::cout << "Setting to AUTOMATIC" << std::endl;
            std::cout << "-----------------------------" 
                      << " AUTOMATIC MODE COMPLETED 100% " 
                      << "------------------------------"
                      << std::endl;
            mode = 'a';
        } else 
        {
            std::cout <<"There was an error while parsing the input."
                      << std::endl << input 
                      << " is not a valid mode." 
                      << std::endl
                      << "Try writing:\nm: manual\na: automatic"
                      << std::endl;
            exit(0);
        }

        input = argv[2];
        // --------------------------------------------------------------------
        // ------------------------ PLAYING DIFFICULTY ------------------------
        // --------------------------------------------------------------------
        if(input.compare("1") == 0) 
        {
            std::cout << "Setting to EASY" << std::endl;   
            std::cout << "--------------------------------" 
                      << " EASY MODE COMPLETED 100% " 
                      << "--------------------------------"
                      << std::endl;
            difficulty = 1;
        } else if (input.compare("2") == 0)
        {

            std::cout << "Setting to HARD" << std::endl;   
            std::cout << "--------------------------------" 
                      << " HARD MODE COMPLETED 100% " 
                      << "--------------------------------"
                      << std::endl;
            difficulty = 2;
        } else 
        {
            std::cout << "There was an error while parsing the input." 
                      << std::endl << input 
                      << " is not a valid playing dificulty." 
                      << std::endl
                      << "Try writing:\n1: easy\n2: hard" 
                      << std::endl;
            exit(0);
        }

        // -------------------------------------------------------------------
        // ------------------------ ROWS & COLUMNS ---------------------------
        // -------------------------------------------------------------------
        numRows    = std::stoi(argv[3]);
        numColumns = std::stoi(argv[4]);

        if(numRows < 0)
        {
            std::cout <<"The number of rows is not valid, it should be bigger than 0"
                      << std::endl;
            exit(0);
        }

        if(numColumns < 0)
        {
            std::cout << "The number of columns is not valid, it should be bigger than 0"
                      << std::endl;
            exit(0);
        }

        numMaxThreads = getThreadsBlock();
        
        if(numRows * numColumns > numMaxThreads)
        {
            std::cout << "The board is too big, please specify other board size."
                      << "The maximum number of cells is " << numMaxThreads
                      << std::endl;
            exit(0);
        } 
       
        // EXECUTE GAME.
        playGame(difficulty, numRows, numColumns, numMaxThreads, mode);
    }
}


// -----------------------------------------------------------------------------
// ----------------------------- CUDA METHODS-----------------------------------
// -----------------------------------------------------------------------------
hipError_t cellsMerge(char movement, int row, int column, int* matrix, 
                       int* POINTS, int* CELLS_OCCUPIED)
{
    // Variable to operate in the GPU
    int* dev_matrix = 0;
    int* dev_POINTS = 0;
    int* dev_CELLSO = 0;

    // TODO: Set Kernel dimensions correctly, probably need a TILE_WIDTH
    // GPU threads distribution
    dim3 dimGrid(row, column, 1);
    dim3 dimBlock(1, 1);

    // Selection of the GPU were code is to be executed
    hipSetDevice(0);
    check_CUDA_Error("hipSetDevice failed!\n");
    
    // Memory Allocation for GPU Variables
    hipMalloc((void**) &dev_matrix, row * column * sizeof(int));
    check_CUDA_Error("hipMalloc failed at Matrix!\n");

    hipMalloc((void**) &dev_POINTS, sizeof(int));
    check_CUDA_Error("hipMalloc failed at POINTS!\n");

    hipMalloc((void**) &dev_CELLSO, sizeof(int));
    check_CUDA_Error("hipMalloc failed at CELLS_OCCUPIED!\n");

    // Memory Transfer: CPU -> GPU
    hipMemcpy(dev_matrix, matrix, row * column * sizeof(int),
               hipMemcpyHostToDevice);
    check_CUDA_Error("cudaMemCpy failed at Matrix (CPU -> GPU)!\n");
    
    hipMemcpy(dev_POINTS, POINTS, sizeof(int),
               hipMemcpyHostToDevice);
    check_CUDA_Error("cudaMemCpy failed at POINTS (CPU -> GPU)!\n");
    
    hipMemcpy(dev_CELLSO, CELLS_OCCUPIED, sizeof(int),
               hipMemcpyHostToDevice);
    check_CUDA_Error("cudaMemCpy failed at CELLS_OCCUPIED (CPU -> GPU)!\n");
    
    /*
     * If the movement is UP or DOWN:
     *     The number of threads is the number of columns.
     * If the movement is LEFT or RIGHT 
     *     The number of threads is the number fo rows.
     */
    switch(movement)
    {
        case 'w':
            fillSpace<<<1, column>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");

            computeMatrixUp<<<1, column>>>(row, column, dev_matrix, 
                                           dev_POINTS, dev_CELLSO);
            check_CUDA_Error("Error merging cells!\n");
            
            fillSpace<<<1, column>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");
            break;

        case 's':
            fillSpace<<<1, column>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");

            computeMatrixDown<<<1, column>>>(row, column, dev_matrix,
                                             dev_POINTS, dev_CELLSO);
            check_CUDA_Error("Error merging cells!\n");
            
            fillSpace<<<1, column>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");
            break;
        
        case 'a':
            fillSpace<<<1, row>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");

            computeMatrixLeft<<<1, row>>>(row, column, dev_matrix, 
                                          dev_POINTS, dev_CELLSO);
            check_CUDA_Error("Error merging cells!\n");
            
            fillSpace<<<1, row>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");
            break;
        
        case 'd':
            fillSpace<<<1, row>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");

            computeMatrixRight<<<1, row>>>(row, column, dev_matrix,
                                           dev_POINTS, dev_CELLSO);
            check_CUDA_Error("Error merging cells!\n");
            
            fillSpace<<<1, row>>>(dev_matrix, movement, row, column);
            check_CUDA_Error("Error while gathering cells!\n");
            break;
    }
    
    // Waits for kernel to finish
    hipDeviceSynchronize();
    check_CUDA_Error("hipDeviceSynchronize returned error!\n");

    // Waits for kernel to finish
    hipDeviceSynchronize();
    check_CUDA_Error("hipDeviceSynchronize returned error!\n");

    // Memory Transfer: GPU -> CPU
    hipMemcpy(matrix, dev_matrix, row * column * sizeof(int),
               hipMemcpyDeviceToHost);
    check_CUDA_Error("cudaMemCpy failed at Matrix (GPU -> CPU)!\n");

    hipMemcpy(POINTS, dev_POINTS, sizeof(int),
               hipMemcpyDeviceToHost);
    check_CUDA_Error("cudaMemCpy failed at POINTS (GPU -> CPU)!\n");

    hipMemcpy(CELLS_OCCUPIED, dev_CELLSO, sizeof(int),
               hipMemcpyDeviceToHost);
    check_CUDA_Error("cudaMemCpy failed at CELLS_OCCUPIED (GPU -> CPU)!\n");
    return hipGetLastError();
}

