// -----------------------------------------------------------------------------
// ------------------------------ LIBRARIES ------------------------------------
// -----------------------------------------------------------------------------
// Includes CUDA
#include <hip/hip_runtime.h>
// CUDA get characteristics

// Allows to implement all string related methods and variables
#include <string>
// C++ library for I/O 
#include <iostream>
// Calloc, exit, free, rand
#include <stdlib.h>
// Time
#include <time.h>

// Size of the tile to be used
#define TILE_WIDTH 1

// Number which allows to know if the matrix is fully occupied
int cellsOccupied = 0;

// -----------------------------------------------------------------------------
// ------------------------------- HEADERS -------------------------------------
// -----------------------------------------------------------------------------
hipError_t sendMatrixToGpu(char movement, int row, int column, int* matrix);

// -----------------------------------------------------------------------------
// ------------------------------- KERNELS -------------------------------------
// -----------------------------------------------------------------------------
// TODO
/*__global__ computeMatrixUp(int rows, int columns, int* matrix)
{
    // Matrix dimensions
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    // Location in matrix
    int row =    by * TILE_WIDTH + ty;
    int column = bx * TILE_WIDTH + tx;

    if(row < rows && column < columns)
    {
        
    }

}
*/

/*
 * Fills the empty spaces in the matrix
 *
 * matrix: The matrix which is going to be filled
 * movement: Movement performed by the user (w, s, a, d)
 */
__global__ void fillSpace(int* matrix, char movement, int rows, int columns)
{
    // Matrix dimensions
    int bx = blockIdx.x;  
    int tx = threadIdx.x; 

    // Location in matrix
    int pos = bx * TILE_WIDTH + tx;
    
    switch(movement)
    {
        // Up
        case 'w':
            // From last row to first row
            for(int i = rows - 1; i > 0; i--)
            {
                for(int j = rows - 1; j > 0; j--)
                {
                    // Current cell NOT 0 and upper cell IS 0 ->
                    // moves current cell up
                    if(matrix[j *     rows + pos] != 0 &&
                       matrix[(j - 1) * rows + pos] == 0)
                    {
                        matrix[(j - 1) * rows + pos] = matrix[j * rows + pos];
                        matrix[j * rows + pos] = 0;
                    }
                }
            }
            break;

        // Down
        case 's':
            // From first row to last row
            for(int i = 0; i < rows; i++)
            {
                for(int j = 0; j < rows - 1; j++)
                {
                    // Current cell NOT 0 and lower cell IS 0 ->
                    // moves current cell down
                    if(matrix[j *     rows + pos] != 0 &&
                       matrix[(j + 1) * rows + pos] == 0)
                    {
                        matrix[(j + 1) * rows + pos] = matrix[j * rows + pos];
                        matrix[j * rows + pos] = 0;
                    }
                }
            }
            break;

        // Left
        case 'a':
            // From last row to first row
            for(int i = columns - 1; i > 0; i--)
            {
                for(int j = columns - 1; j > 0; j--)
                {
                    // Current cell NOT 0 and upper cell IS 0 ->
                    // moves current cell up
                    if(matrix[pos * rows +  j]      != 0 &&
                       matrix[pos * rows + (j - 1)] == 0)
                    {
                        matrix[pos * rows + (j - 1)] = matrix[pos * rows + j];
                        matrix[pos * rows +  j] = 0;
                    }
                }
            }
            
            break;

        //  Right
        case 'd':
            // From first row to last row
            for(int i = 0; i < columns; i++)
            {
                for(int j = 0; j < columns - 1; j++)
                {
                    // Current cell NOT 0 and lower cell IS 0 ->
                    // moves current cell down
                    if(matrix[pos * rows + j]       != 0 &&
                       matrix[pos * rows + (j + 1)] == 0)
                    {
                        matrix[pos * rows + (j + 1)] = matrix[pos * rows + j];
                        matrix[pos * rows +  j] = 0;
                    }
                }
            }
            break;
    }

}

// -----------------------------------------------------------------------------
// -------------------------- CUDA RELATED METHODS -----------------------------
// -----------------------------------------------------------------------------
// Method which allows to check for errors
__host__ void check_CUDA_Error(const char *msg)
{
    hipError_t  err;

    hipDeviceSynchronize();

    err = hipGetLastError();

    if(err != hipSuccess)
    {
        std::cerr << "ERROR " << err << " OCURRED: " << hipGetErrorString(err)  
                  << "(" << msg << ")" << std::endl;
        std::cerr << "Press any key to finish execution..." << std::endl;
        fflush(stdin);

        char key = getchar();

        exit(-1);
    }
}

// Gets the number of threads allowed per block in the current GPU
__host__ int getThreadsBlock()
{
    int threadsBlock = 0;

    int nDevices;

    hipGetDeviceCount(&nDevices);
    check_CUDA_Error("Couldn't get the number of devices in this computer");

    if(nDevices > 0)
    {
        hipDeviceProp_t prop;
        
        hipGetDeviceProperties(&prop, 0);
        
        threadsBlock = prop.maxThreadsPerBlock;
    }

    return threadsBlock;
}

// -----------------------------------------------------------------------------
// ------------------------------ GAME METHODS ---------------------------------
// -----------------------------------------------------------------------------
__host__ void displayGrid(int rows, int columns, int* Matrix)
{
    system("clear");

    /*std::cout << 
        "--------------------------------------------------------------------------------"
        << std::endl;
    std::cout << "16384" << std::endl;    
    std::cout << 
        "--------------------------------------------------------------------------------"
        << std::endl << std::endl;
    */
    // Two extra iterations to print the upper part of the matrix
    for(int i = -2; i < rows; i++)
    {
        if(i < 0) {
            std::cout << "      ";
        } else if(i + 1 < 10) {
            std::cout << i + 1 << " - | ";
        } else if(i + 1 >= 10) {
            std::cout << i + 1 << "- | ";
        }

        for(int j = 0; j < columns; j++)
        {
            if(i == -2) {
                if(j + 1 < 10) {
                    std::cout << j + 1 << "   ";
                } else {
                    std::cout << j + 1 << "  ";
                }
            } else if(i == -1) {
                std::cout << "|   "; 
            } else  {
                std::cout << Matrix[i * rows + j] << " | ";
            }
        }

        std::cout << std::endl;
    }

    std::cout << std::endl;

    std::cout <<                          std::endl 
              <<                          std::endl 
              << "Controls: "          << std::endl
              << "        ___"         << std::endl
              << "       | W |"        << std::endl
              << " ___    ___    ___"  << std::endl
              << "| A |  | S |  | D |" << std::endl
              <<                          std::endl;
}

__host__ void seeding(int gameDifficulty, int rows, int columns, int* matrix)
{
    // Number of seeds to be planted in the board
    int seeds;
    // Values the seeds might have while inserted
    int* seedsValues;
    // Number of seeds planted 
    int seedsPlanted = 0;
    // Position from the matrix where the seed is going to be planted. 
    // Used auxiliary variable
    int position;

    // Depending on the game difficulty, the number of seeds may vary
    switch(gameDifficulty)
    {
        case 1:
            seeds = 15;
            seedsValues = (int*) calloc(3, sizeof(int));
            seedsValues[0] = 2;
            seedsValues[1] = 4;
            seedsValues[2] = 8;
            break;

        case 2:
            seeds = 8;
            seedsValues = (int*) calloc(2, sizeof(int));
            seedsValues[0] = 2;
            seedsValues[1] = 4;
            break;
    }

    // Initialize random seed
    std::srand(time(0));

    while(seedsPlanted < seeds)
    {
        // Still empty cells
        if(cellsOccupied < (rows * columns - 1))
        {
        // Position within the matrix
        position = rand() % ((rows * columns) - 1);

        if(matrix[position] == 0)
        {
                // Random seed value among the ones according to the difficulty
                matrix[position] = seedsValues[rand() % 
                                               (sizeof(seedsValues) 
                                                / sizeof(int) 
                                                - 1)];
                seedsPlanted++;
            }
        } 
        else
        {
            std::cout << "There are no more empty cells, you have lost a live"  << std::endl;
            // TODO:
            // Probably it would be needed decrease the number of lives
            // Therefore, it is needed a new method for that, called here
        }

    }

}

// -----------------------------------------------------------------------------
// -------------------------------- MAIN CODE ----------------------------------
// -----------------------------------------------------------------------------
int main(int argc, char** argv)
{
    // Game Mode
    char mode;
    // Game Difficulty
    int  difficulty;
    // Board Height
    int  numRows;
    // Board Weight
    int  numColumns;
    // Board Maximum Number of Cells
    int  numMaxThreads;  
    // Used as auxiliary variable for any input in the system
    std::string input;

    system("clear");
    std::cout << "Processing game settings" << std::endl;

    if(argc == 5)
    {
        input = argv[1];
        
        // --------------------------------------------------------------------
        // --------------------------- PLAYING MODE ---------------------------
        // --------------------------------------------------------------------
        if(input.compare("m")) 
        {
            std::cout << "Setting to MANUAL" << std::endl;   
            std::cout << "-------------------------------" 
                      << " MANUAL MODE COMPLETED 100% " 
                      << "-------------------------------"
                      << std::endl;
            mode = 'm';
        } else if (input.compare("a"))
        {
            std::cout << "Setting to AUTOMATIC" << std::endl;
            std::cout << "-----------------------------" 
                      << " AUTOMATIC MODE COMPLETED 100% " 
                      << "------------------------------"
                      << std::endl;
            mode = 'a';
        } else 
        {
            std::cout <<"There was an error while parsing the input."
                      << std::endl << input 
                      << " is not a valid mode." 
                      << std::endl
                      << "Try writing:\nm: manual\na: automatic"
                      << std::endl;
            exit(0);
        }

        input = argv[2];
        // --------------------------------------------------------------------
        // ------------------------ PLAYING DIFFICULTY ------------------------
        // --------------------------------------------------------------------
        if(input.compare("1")) 
        {
            std::cout << "Setting to EASY" << std::endl;   
            std::cout << "--------------------------------" 
                      << " EASY MODE COMPLETED 100% " 
                      << "--------------------------------"
                      << std::endl;
            difficulty = 1;
        } else if (input.compare("2"))
        {

            std::cout << "Setting to HARD" << std::endl;   
            std::cout << "--------------------------------" 
                      << " HARD MODE COMPLETED 100% " 
                      << "--------------------------------"
                      << std::endl;
            difficulty = 2;
        } else 
        {
            std::cout << "There was an error while parsing the input." 
                      << std::endl << input 
                      << " is not a valid playing dificulty." 
                      << std::endl
                      << "Try writing:\n1: easy\n2: hard" 
                      << std::endl;
            exit(0);
        }

        // -------------------------------------------------------------------
        // ------------------------ ROWS & COLUMNS ---------------------------
        // -------------------------------------------------------------------
        numRows    = std::stoi(argv[3]);
        numColumns = std::stoi(argv[4]);

        if(numRows < 0)
        {
            std::cout <<"The number of rows is not valid, it should be bigger than 0"
                      << std::endl;
        }

        if(numColumns < 0)
        {
            std::cout << "The number of columns is not valid, it should be bigger than 0"
                      << std::endl;
        }

        numMaxThreads = getThreadsBlock();
        
        if(numRows * numColumns > numMaxThreads)
        {
            std::cout << "The board is too big, please specify other board size."
                      << "The maximum number of cells is " << numMaxThreads
                      << std::endl;
            exit(0);
        } 
    
        int* Matrix = (int*) calloc(numRows * numColumns, sizeof(int));
     
        bool play;

        seeding(difficulty, numRows, numColumns, Matrix);
        displayGrid(numRows, numColumns, Matrix);
        
        do {
            std::cin >> input;

            if(input.length() == 1) {
                sendMatrixToGpu(input[0], numRows, numColumns, Matrix);
                displayGrid(numRows, numColumns, Matrix);
            } else {
                std::cout << "not that one cracker!" << std::endl;
            }
        } while(play);

    }
}

// -----------------------------------------------------------------------------
// ----------------------------- CUDA METHODS-----------------------------------
// -----------------------------------------------------------------------------
hipError_t sendMatrixToGpu(char movement, int row, int column, int* matrix)
{
    // Variable to operate in the GPU
    int* dev_matrix = 0;
    
    // TODO: Set Kernel dimensions correctly, probably need a TILE_WIDTH
    // GPU threads distribution
    dim3 dimGrid(row, column, 1);
    dim3 dimBlock(1, 1);

    // Selection of the GPU were code is to be executed
    hipSetDevice(0);
    check_CUDA_Error("hipSetDevice failed!\n");
    
    // Memory Allocation for GPU Variables
    hipMalloc((void**) &dev_matrix, row * column * sizeof(int));
    check_CUDA_Error("hipMalloc failed at Matrix!\n");

    // Memory Transfer: CPU -> GPU
    hipMemcpy(dev_matrix, matrix, row * column * sizeof(int),
               hipMemcpyHostToDevice);
    check_CUDA_Error("cudaMemCpy failed at Matrix!\n");
    
    /*
     * If the movement is UP or DOWN:
     *     The number of threads is the number of columns.
     * If the movement is LEFT or RIGHT 
     *     The number of threads is the number fo rows.
     */
    if(movement == 'w' || movement == 's')
    {
        fillSpace<<<1, column>>>(dev_matrix, movement, row, column);
    } 
    else if(movement == 'a' || movement == 'd')
    {
        fillSpace<<<1, row>>>(dev_matrix, movement, row, column);
    }
    check_CUDA_Error("Error while gathering cells\n");

    // Waits for kernel to finish
    hipDeviceSynchronize();
    check_CUDA_Error("hipDeviceSynchronize returned error!");

    // Computes the matrix joining the numbers with the same values
    //computeMatrixUp<<<, >>>(rows, columns, dev_matrix);   
    //check_CUDA_Error("Error after trying to mix cells!\n");

    // Waits for kernel to finish
    hipDeviceSynchronize();
    check_CUDA_Error("hipDeviceSynchronize returned error!");

    // Memory Transfer: GPU -> CPU
    hipMemcpy(matrix, dev_matrix, row * column * sizeof(int),
               hipMemcpyDeviceToHost);
    check_CUDA_Error("cudaMemCpy failed after copying from GPU to CPU!\n");

    return hipGetLastError();
}

