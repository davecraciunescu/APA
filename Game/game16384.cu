// Includes CUDA
#include <hip/hip_runtime.h>
// CUDA get characteristics

// Allows to implement all string related methods and variables
#include <string>
// C++ library for I/O 
#include <iostream>

// Method which allows to check for errors
__host__ void check_CUDA_Error(const char *msg)
{
    hipError_t  err;

    hipDeviceSynchronize();

    err = hipGetLastError();

    if(err != hipSuccess)
    {
        std::cerr << "ERROR " << err << "OCURRED: " << hipGetErrorString(err)  
                  << "(" << msg << ")" << std::endl;
        std::cerr << "Press any key to finish execution..." << std::endl;
        fflush(stdin);

        char key = getchar();

        exit(-1);
    }
}

// Gets the number of threads allowed per block in the current GPU
__host__ int getThreadsBlock()
{
    int threadsBlock = 0;

    int nDevices;

    hipGetDeviceCount(&nDevices);
    check_CUDA_Error("Couldn't get the number of devices in this computer");

    for(int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        
        hipGetDeviceProperties(&prop, i);
        
        threadsBlock = prop.maxThreadsPerBlock;
    }

    return threadsBlock;
}

int main(int argc, char** argv)
{
    // Game Mode
    std::string mode;
    // Game Difficulty
    std::string dificulty;
    // Board Height
    int         numRows;
    // Board Weight
    int         numColumns;
    // Board Maximum Number of Cells
    int         numMaxThreads;  
    
    system("clear");
    std::cout << "Processing game settings" << std::endl;

    if(argc == 5)
    {
        std::string input = argv[1];
        
        // --------------------------------------------------------------------
        // --------------------------- PLAYING MODE ---------------------------
        // --------------------------------------------------------------------
        if(input.compare("m")) 
        {
            std::cout << "Setting to MANUAL" << std::endl;   
            std::cout << "-------------------------------" 
                      << " MANUAL MODE COMPLETED 100% " 
                      << "-------------------------------"
                      << std::endl;
        } else if (input.compare("a"))
        {
            std::cout << "Setting to AUTOMATIC" << std::endl;
            std::cout << "-----------------------------" 
                      << " AUTOMATIC MODE COMPLETED 100% " 
                      << "------------------------------"
                      << std::endl;
        } else 
        {
            std::cout <<"There was an error while parsing the input."
                      << std::endl << input 
                      << " is not a valid mode." 
                      << std::endl
                      << "Try writing:\nm: manual\na: automatic"
                      << std::endl;
        }

        input = argv[2];
        // --------------------------------------------------------------------
        // ------------------------ PLAYING DIFFICULTY ------------------------
        // --------------------------------------------------------------------
        if(input.compare("1")) 
        {
            std::cout << "Setting to EASY" << std::endl;   
            std::cout << "--------------------------------" 
                      << " EASY MODE COMPLETED 100% " 
                      << "--------------------------------"
                      << std::endl;
            
        } else if (input.compare("2"))
        {

            std::cout << "Setting to HARD" << std::endl;   
            std::cout << "--------------------------------" 
                      << " HARD MODE COMPLETED 100% " 
                      << "--------------------------------"
                      << std::endl;
        } else 
        {
            std::cout << "There was an error while parsing the input." 
                      << std::endl << input 
                      << " is not a valid playing dificulty." 
                      << std::endl
                      << "Try writing:\n1: easy\n2: hard" 
                      << std::endl;
        }

        // -------------------------------------------------------------------
        // ------------------------ ROWS & COLUMNS ---------------------------
        // -------------------------------------------------------------------
        numRows    = std::stoi(argv[3]);
        numColumns = std::stoi(argv[4]);

        if(numRows < 0)
        {
            std::cout <<"The number of rows is not valid, it should be bigger than 0"
                      << std::endl;
        }

        if(numColumns < 0)
        {
            std::cout << "The number of columns is not valid, it should be bigger than 0"
                      << std::endl;
        }

        numMaxThreads = getThreadsBlock();
        
        if(numRows * numColumns > numMaxThreads)
        {
            std::cout << "The board is too big, please specify other board size."
                      << "The maximum number of cells is " << numMaxThreads
                      << std::endl;
        }
    }
}

