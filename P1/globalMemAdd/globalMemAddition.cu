#include <hip/hip_runtime.h>

#include <stdio.h>

hipError_t addThreeNumbersCuda(int* result, int* a, int* b, int* c);

__global__ void addThreeNums(int* result, int* a, int* b, int* c)
{
	*result = *a + *b + *c;
}

int main()
{
	int a = 5;
	int b = 10;
	int c = 15;
	int result = 0;

	hipError_t err = addThreeNumbersCuda(&result, &a, &b, &c);
	if (err != hipSuccess) {
		fprintf(stderr, "addThreeNumbersCuda failed!\n");
		return 1;
	}

	printf("%d + %d + %d = %d\n", a, b, c, result);

	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}

hipError_t addThreeNumbersCuda(int* result, int* a, int* b, int* c)
{
	// As many cuda errors are going to be treated, it is better to initialize
	// here the error variable
	hipError_t err;
	// Variables used to operate in GPU mem
	int* dev_result = 0;
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	// Select GPU where the threads are to be executed
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!");
		goto Error;
	}

	// Allocate memory for GPU's variables
	err = hipMalloc((void**)&dev_result, sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_a, sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_b, sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	err = hipMalloc((void**)&dev_c, sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMaloc failed!");
		goto Error;
	}

	// Transfers data from host variables to GPU variables
	err = hipMemcpy(dev_a, a, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	err = hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	err = hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
    
    // Launch kernel to add three values
	addThreeNums <<<1, 1>>>(dev_result, dev_a, dev_b, dev_c);

	// Checks whether the execution in the GPU has been completed correctly
	err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "addThreeNums launch failed: %s\n",
			hipGetErrorString(err));
		goto Error;
	}

	// Waits for the kernel to finish and checks whether theres has been any
	// errors or not
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addThreeNums!\n", err);
		goto Error;
	}

	err = hipMemcpy(result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return err;
}

