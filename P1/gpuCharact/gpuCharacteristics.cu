#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{
	int nDevices;
	// As the function that points also retrieves the hipErrorNoDevice
	// the result must be checked
	hipError_t err = hipGetDeviceCount(&nDevices);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	// Retrives information from every device obtained (if any)
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;

		hipGetDeviceProperties(&prop, i);

		printf("Device Number: %d\n", i);
		printf("    Device Name: %s\n", prop.name);
		printf("    Memory Clock Rate (KHz): %d\n",
			   prop.memoryClockRate);
		printf("    Memory Bus Width (bits): %d\n",
			   prop.memoryBusWidth);
		printf("    Peak Memory Bandwidth (GB/s): %f\n\n",
			// It is obtained at first bits/s, which wants to be transformed into
			// GB/s -> 1/8 and 1/1000000
			   2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}

	return 0;

}
