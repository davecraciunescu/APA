#include <stdio.h>

int main() 
{
    int nDevices;

    hipError_t err = hipGetDeviceCount(&nDevices);
    if(err != hipSuccess) printf("%s\n", hipGetErrorString(err));

    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;

        hipGetDeviceProperties(&prop, i);

        printf("Device Number: %d\n", i);
        printf("    Device Name: %s\n", prop.name);
        printf("    Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("    Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("    Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) 1.0e6);       
    }

    return 0;

}
